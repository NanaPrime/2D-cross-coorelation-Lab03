#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 256
#define KERNEL_SIZE 8
#define OUTPUT_SIZE (INPUT_SIZE - KERNEL_SIZE + 1)

__constant__ float d_kernel[KERNEL_SIZE * KERNEL_SIZE];

// Function to initialize matrix with random values between -1 and 1
void initializeMatrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = ((float)rand() / RAND_MAX) * 2.0f - 1.0f;  // Random float between -1 and 1
    }
}

// CUDA Kernel for 2D cross-correlation
__global__ void crossCorrelateKernel(float* input, float* output) {
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_x < OUTPUT_SIZE && out_y < OUTPUT_SIZE) {
        float sum = 0.0f;
        for (int m = 0; m < KERNEL_SIZE; m++) {
            for (int n = 0; n < KERNEL_SIZE; n++) {
                sum += input[(out_y + m) * INPUT_SIZE + (out_x + n)] * d_kernel[m * KERNEL_SIZE + n];
            }
        }
        output[out_y * OUTPUT_SIZE + out_x] = sum;
    }
}

int main() {
    // Host matrices
    float* h_input = (float*)malloc(INPUT_SIZE * INPUT_SIZE * sizeof(float));
    float* h_kernel = (float*)malloc(KERNEL_SIZE * KERNEL_SIZE * sizeof(float));
    float* h_output = (float*)malloc(OUTPUT_SIZE * OUTPUT_SIZE * sizeof(float));

    // Initialize random seed
    srand((unsigned int)time(NULL));

    // Initialize matrices on the host
    initializeMatrix(h_input, INPUT_SIZE, INPUT_SIZE);
    initializeMatrix(h_kernel, KERNEL_SIZE, KERNEL_SIZE);

    // Device memory allocation
    float *d_input, *d_output;
    hipMalloc(&d_input, INPUT_SIZE * INPUT_SIZE * sizeof(float));
    hipMalloc(&d_output, OUTPUT_SIZE * OUTPUT_SIZE * sizeof(float));

    // Copy the input matrix to device memory
    hipMemcpy(d_input, h_input, INPUT_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
    
    // Copy the kernel matrix to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(float));

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the kernel
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((OUTPUT_SIZE + 15) / 16, (OUTPUT_SIZE + 15) / 16);
    crossCorrelateKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output);

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CUDA Kernel Execution Time: %.4f ms\n", milliseconds);

    // Copy the result back to the host
    hipMemcpy(h_output, d_output, OUTPUT_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Print the output matrix (top-left 4x4 elements)
    printf("Output (Top 4x4):\n");
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            printf("%6.2f ", h_output[i * OUTPUT_SIZE + j]);
        }
        printf("\n");
    }

    // Cleanup
    free(h_input);
    free(h_kernel);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
